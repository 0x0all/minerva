#include "hip/hip_runtime.h"
#include "op/impl/cuda/cuda_kernel.h"
#include "op/impl/cuda/cuda_perform.h"
#include "common/cuda_utils.h"
#include <glog/logging.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <limits>
#include "stdio.h"

namespace minerva {
namespace cuda {

static void FindConfiguration(size_t size, int& num_blocks, int& num_threads, bool block_crop = true) {
  num_threads = size < 1024? 256 : 1024;
  num_blocks = static_cast<int>((size + num_threads - 1) / num_threads);
  if (num_blocks < 0 || 128 < num_blocks) {
    num_blocks = 128;
  }
  //printf("#s=%d #t=%d #b=%d\n", size, num_threads, num_blocks);
  
  /*num_threads = 32;
  num_blocks = static_cast<int>((size + num_threads - 1) / num_threads);
  if (num_blocks < 0 || 128 < num_blocks) {
    num_blocks = 128;
  }*/
}

void CudaPerformDotMult(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, MultOp());
  CheckCudaError("CudaPerformDotMult");
}

void CudaPerformDotDiv(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, DivOp());
  CheckCudaError("CudaPerformDotDiv");
}

void CudaPerformAdd(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, SumOp());
  CheckCudaError("CudaPerformAdd");
  //float one = 1.0;
  //CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  //CUBLAS_CALL(hipblasSaxpy(handle, size, &one, b, 1, c, 1));
}

void CudaPerformCopy(float* a, float* b, size_t size, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, b, 1));
}

void CudaPerformSub(float* a, float* b, float* c, size_t size, hipblasHandle_t handle) {
  float minus_one = -1.0;
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  CUBLAS_CALL(hipblasSaxpy(handle, size, &minus_one, b, 1, c, 1));
}

void CudaPerformMatMult(float* a, float* b, float* c, int m, int n, int k, hipblasHandle_t handle) {
  float one = 1.0;
  float zero = 0.0;
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, a, m, b, k, &zero, c, m));
}

void CudaPerformScale(float* in_data, float* res_data, size_t size, float val, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, in_data, 1, res_data, 1));
  CUBLAS_CALL(hipblasSscal(handle, size, &val, res_data, 1));
}

void CudaPerformTranspose(float* a, float* c, int m, int n, hipblasHandle_t handle) {
  float zero = 0.0;
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &one, a, m, &zero, c, n, c, n));
}

void CudaPerformConstAdd(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, SumOp());
  CheckCudaError("CudaPerformConstAdd");
}

void CudaPerformLeftConstSub(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseSubOp());
  CheckCudaError("CudaPerformLeftConstSub");
}

void CudaPerformLeftConstDiv(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseDivOp());
  CheckCudaError("CudaPerformLeftConstDiv");
}

void CudaPerformNormAddOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnCol");
}

void CudaPerformNormSubOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnCol");
}

void CudaPerformNormMultOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnCol");
}

void CudaPerformNormDivOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnCol");
}

void CudaPerformNormAddOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnRow");
}

void CudaPerformNormSubOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnRow");
}

void CudaPerformNormMultOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnRow");
}

void CudaPerformNormDivOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnRow");
}

void CudaPerformReductionSumOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnCol");
}

void CudaPerformReductionMaxOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnCol");
}

void CudaPerformReductionSumOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnRow");
}

void CudaPerformReductionMaxOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnRow");
}

void CudaPerformMaxIndexOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformMaxIndexOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnCol");
}

void CudaPerformMaxIndexOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformMaxIndexOnRowKernel << <block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnRow");
}

void CudaPerformReshape(float* in, float* out, size_t size, hipStream_t stream) {
  CUDA_CALL(hipMemcpyAsync(out, in, size, hipMemcpyDefault, stream));
}

void CudaPerformElewiseExp(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, ExpOp());
  CheckCudaError("CudaPerformEleWiseExp");
}

void CudaPerformElewiseLn(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, LnOp());
  CheckCudaError("CudaPerformEleWiseLn");
}

void CudaPerformElewiseNegative(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, NegativeOp());
  CheckCudaError("CudaPerformEleWiseNegative");
}

void CudaPerformConvForward(float* bottom, float* filter, float* bias, float* top, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_desc;
  cudnnTensor4dDescriptor_t bias_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bias_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_desc, filter_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  float one = 1;
  CUDNN_CALL(hipdnnConvolutionForward(handle, bottom_desc, bottom, filter_desc, filter, conv_desc, top_desc, top, CUDNN_RESULT_NO_ACCUMULATE));
  CUDNN_CALL(cudnnAddTensor4d(handle, CUDNN_ADD_SAME_C, &one, bias_desc, bias, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bias_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformConvBackwardData(float* top_diff, float* filter, float* bottom_diff, int num_images, int bottom_num_channels, int top_num_channels, int top_height, int top_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_diff_desc;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_diff_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, (top_height - 1) * stride_vertical + filter_height - 2 * pad_height, (top_width - 1) * stride_horizontal + filter_width - 2 * pad_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_diff_desc, filter_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  CUDNN_CALL(hipdnnConvolutionBackwardData(handle, filter_desc, filter, top_diff_desc, top_diff, conv_desc, bottom_diff_desc, bottom_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_diff_desc));
}

void CudaPerformConvBackwardFilter(float* bottom, float* top_diff, float* filter_diff, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_diff_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_diff_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_diff_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_desc, filter_diff_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle, bottom_desc, bottom, top_diff_desc, top_diff, conv_desc, filter_diff_desc, filter_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_diff_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformConvBackwardBias(float* top_diff, float* bias_diff, int num_images, int top_num_channels, int top_height, int top_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bias_diff_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bias_diff_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  CUDNN_CALL(hipdnnConvolutionBackwardBias(handle, top_diff_desc, top_diff, bias_diff_desc, bias_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bias_diff_desc));
}

void CudaPerformInstanceSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformChannelSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformInstanceSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, desc, top, desc, diff, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformChannelSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, desc, top, desc, diff, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformSigmoidForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_SIGMOID, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformReluForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_RELU, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformTanhForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_TANH, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformSigmoidBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_SIGMOID, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformReluBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_RELU, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformTanhBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_TANH, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformMaxPoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  //Calculate the dimension after pooling	
  int pooled_height = static_cast<int>(ceil(static_cast<float>((bottom_height + 2 * pad_height - window_height)) / stride_vertical)) + 1;
  int pooled_width = static_cast<int>(ceil(static_cast<float>((bottom_width + 2 * pad_width - window_width)) / stride_horizontal)) + 1;
  //printf("btm_h=%d btm_w=%d pooled_h=%d pooled_width=%d\n", bottom_height, bottom_width, pooled_height, pooled_width);

  if (pad_height > 0 || pad_width > 0) {
    // has padding, call caffe's pooling mthod
	  if((pooled_height - 1) * stride_vertical >= bottom_height + pad_height)
		--pooled_height;  
	  if((pooled_width - 1) * stride_horizontal >= bottom_width + pad_width)
		--pooled_width;

  	int block, thread;
	  int size = num_images * num_channels * pooled_width * pooled_height;
	  FindConfiguration(size, block, thread, false);
    //printf("size=%d block=%d thread=%d\n", size, block, thread);
	  CudaMaxPoolForward<<<block, thread, 0, stream>>>(
      size, bottom, num_images, num_channels, bottom_height, bottom_width, pooled_height, pooled_width,
      window_height, window_width, stride_vertical, stride_horizontal, pad_height, pad_width, top);
	  CheckCudaError("CudaMaxPoolForward");
  } else {
    // no padding, just call cudnn
    cudnnTensor4dDescriptor_t bottom_desc;
    hipdnnPoolingDescriptor_t pool_desc;
    cudnnTensor4dDescriptor_t top_desc;

    CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
    CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
    CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, stride_vertical, stride_horizontal));
    
    //TODO: Even the formular with ceil is not strictly correct, we need to assure the last pooling starts within the image
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, static_cast<int>(ceil(static_cast<float>((bottom_height - window_height)) / stride_vertical)) + 1, static_cast<int>(ceil(static_cast<float>((bottom_width - window_width)) / stride_horizontal)) + 1));

    CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, bottom_desc, bottom, top_desc, top));
    CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

    CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
    CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
  }
}

void CudaPerformAveragePoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {

  //TODO: haven't implement pad>0 ave pooling
  CHECK_EQ(pad_height, 0) << "pad > 0 ave pooling haven't been implemented"; 
  CHECK_EQ(pad_width, 0) << "pad > 0 ave pooling haven't been implemented"; 
  

  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, CUDNN_POOLING_AVERAGE, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, static_cast<int>(ceil(static_cast<float>((bottom_height - window_height)) / stride_vertical)) + 1, static_cast<int>(ceil(static_cast<float>((bottom_width - window_width)) / stride_horizontal)) + 1));

  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, bottom_desc, bottom, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformMaxPoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

    //Calculate the dimension after pooling	
  int pooled_height = static_cast<int>(ceil(static_cast<float>((bottom_height + 2 * pad_height - window_height)) / stride_vertical)) + 1;
  int pooled_width = static_cast<int>(ceil(static_cast<float>((bottom_width + 2 * pad_width - window_width)) / stride_horizontal)) + 1;

  if (pad_height > 0 || pad_width > 0) {
	  if((pooled_height - 1) * stride_vertical >= bottom_height + pad_height)
		--pooled_height;  
	  if((pooled_width - 1) * stride_horizontal >= bottom_width + pad_width)
		--pooled_width;

	  int block, thread;
	  int size = num_images * num_channels * bottom_width * bottom_height;
	  FindConfiguration(size, block, thread, false);

	  //set bottom_diff 0
	  CudaPerformFillKernel<<<block, thread, 0, stream>>>(bottom_diff, size, 0.0);
  	CheckCudaError("CudaPerformFill");
	  
	  CudaMaxPoolBackward<<<block, thread, 0, stream>>>(
	  size, bottom, top_diff, bottom_diff, num_images, num_channels, bottom_height, bottom_width, pooled_height, pooled_width, window_height, window_width, stride_vertical, stride_horizontal, pad_height, pad_width);
	  CheckCudaError("CudaMaxPoolBackward");
  } else {
    CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
    CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
    CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, stride_vertical, stride_horizontal));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, static_cast<int>(ceil(static_cast<float>((bottom_height - window_height)) / stride_vertical)) + 1, static_cast<int>(ceil(static_cast<float>((bottom_width - window_width)) / stride_horizontal)) + 1));

    CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, top_desc, top, top_desc, top_diff, bottom_desc, bottom, bottom_desc, bottom_diff));
    CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

    CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
    CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
  }
}

void CudaPerformAveragePoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, CUDNN_POOLING_AVERAGE, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, static_cast<int>(ceil(static_cast<float>((bottom_height - window_height)) / stride_vertical)) + 1, static_cast<int>(ceil(static_cast<float>((bottom_width - window_width)) / stride_horizontal)) + 1));

  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, top_desc, top, top_desc, top_diff, bottom_desc, bottom, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformRandn(float* dst, size_t size, unsigned int seed, float mean, float var) {
  hiprandGenerator_t gen;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
  CURAND_CALL(hiprandGenerateNormal(gen, dst, size, mean, var));
  CURAND_CALL(hiprandDestroyGenerator(gen));
}

void CudaPerformRandBernoulli(float* dst, size_t size, unsigned int seed, float p, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformRandBernoulliKernel<<<block, thread, 0, stream>>>(dst, size, seed, p);
  CheckCudaError(__func__);
}

void CudaPerformFill(float* dst, size_t size, float val, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformFillKernel<<<block, thread, 0, stream>>>(dst, size, val);
  CheckCudaError("CudaPerformFill");
}

void CudaPerformLRNForward(float* bottom, float* scale, float* res, int local_size, float alpha, float beta, int num_img, int channel, int width, int height, hipStream_t stream)
{
	int block, thread, size;
	size = num_img * height * width;
	FindConfiguration(size, block, thread, false);
	LRNFillScale<<<block, thread, 0, stream>>>(
    size, bottom, num_img, channel, height, width, local_size,
    alpha / local_size, scale);
	CheckCudaError("LRNFillScale");
	
	size = num_img * channel * width * height;
	FindConfiguration(size, block, thread, false);
	// NOLINT_NEXT_LINE(whitespace/operators)
	LRNComputeOutput<<<block, thread, 0, stream>>>(size, bottom, scale, -beta, res);
	CheckCudaError("LRNComputeOutput");
}

void CudaPerformLRNBackward(float* bottom_data, float* top_data, float* scale, float* top_diff, float* bottom_diff, int local_size, float alpha, float beta, int num_img, int channel, int width, int height, hipStream_t stream)
{
	int block, thread;
	int size = num_img * width * height;
	FindConfiguration(size, block, thread, false);
	LRNComputeDiff<<<block, thread, 0, stream>>>(
    size, bottom_data, top_data, scale, top_diff,  num_img, channel, height, width, local_size,
    -beta, float(2. * alpha * beta / local_size), bottom_diff);
	CheckCudaError("LRNBackward");
}



}  // namespace cuda
}  // namespace minerva

