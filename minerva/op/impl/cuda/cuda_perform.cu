#include "hip/hip_runtime.h"
#include "op/impl/cuda/cuda_perform.h"

__global__ static void CudaPerformArithmeticAddKernel(float* res, float* left, float* right, size_t size) {
  int cur = threadIdx.x + blockIdx.x * blockDim.x;
  while (cur < size) {
    *(res + cur) = *(left + cur) + *(right + cur);
    cur += gridDim.x * blockDim.x;
  }
}

__global__ static void CudaPerformArithmeticSubKernel(float* res, float* left, float* right, size_t size) {
  int cur = threadIdx.x + blockIdx.x * blockDim.x;
  while (cur < size) {
    *(res + cur) = *(left + cur) - *(right + cur);
    cur += gridDim.x * blockDim.x;
  }
}

__global__ static void CudaPerformArithmeticMultKernel(float* res, float* left, float* right, size_t size) {
  int cur = threadIdx.x + blockIdx.x * blockDim.x;
  while (cur < size) {
    *(res + cur) = *(left + cur) * *(right + cur);
    cur += gridDim.x * blockDim.x;
  }
}

__global__ static void CudaPerformArithmeticDivKernel(float* res, float* left, float* right, size_t size) {
  int cur = threadIdx.x + blockIdx.x * blockDim.x;
  while (cur < size) {
    *(res + cur) = *(left + cur) / *(right + cur);
    cur += gridDim.x * blockDim.x;
  }
}

namespace minerva {
namespace cuda {

void CudaPerformArithmeticAdd(float* res, float* left, float* right, size_t size, hipStream_t stream) {
  CudaPerformArithmeticAddKernel<<<16, 16, 0, stream>>>(res, left, right, size);
}

void CudaPerformArithmeticSub(float* res, float* left, float* right, size_t size, hipStream_t stream) {
  CudaPerformArithmeticSubKernel<<<16, 16, 0, stream>>>(res, left, right, size);
}

void CudaPerformArithmeticMult(float* res, float* left, float* right, size_t size, hipStream_t stream) {
  CudaPerformArithmeticMultKernel<<<16, 16, 0, stream>>>(res, left, right, size);
}

void CudaPerformArithmeticDiv(float* res, float* left, float* right, size_t size, hipStream_t stream) {
  CudaPerformArithmeticDivKernel<<<16, 16, 0, stream>>>(res, left, right, size);
}

}
}
