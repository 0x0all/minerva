#include "op/impl/cuda/cuda_kernel.h"
#include "op/impl/cuda/cuda_perform.h"
#include "common/cuda_utils.h"
#include <glog/logging.h>
#include <hipblas.h>
#include <limits>

namespace minerva {
namespace cuda {

static void FindConfiguration(size_t size, int& num_blocks, int& num_threads) {
  num_threads = 32;
  num_blocks = static_cast<int>((size + num_threads - 1) / num_threads);
  if (num_blocks < 0 || 128 < num_blocks) {
    num_blocks = 128;
  }
}

void CudaPerformDotMult(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, MultOp());
  CheckCudaError("CudaPerformDotMult");
}

void CudaPerformDotDiv(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, DivOp());
  CheckCudaError("CudaPerformDotDiv");
}

void CudaPerformAdd(float* a, float* b, float* c, int m, int n, hipblasHandle_t handle) {
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, a, m, &one, b, m, c, m));
}

void CudaPerformSub(float* a, float* b, float* c, int m, int n, hipblasHandle_t handle) {
  float minus_one = -1.0;
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, a, m, &minus_one, b, m, c, m));
}

void CudaPerformMatMult(float* a, float* b, float* c, int m, int n, int k, hipblasHandle_t handle) {
  float one = 1.0;
  float zero = 0.0;
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, a, m, b, k, &zero, c, m));
}

void CudaPerformScale(float* a, float* c, int m, int n, float val, hipblasHandle_t handle) {
  float zero = 0.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &val, a, m, &zero, c, m, c, m));
}

void CudaPerformTranspose(float* a, float* c, int m, int n, hipblasHandle_t handle) {
  float zero = 0.0;
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &one, a, m, &zero, c, n, c, n));
}

void CudaPerformLeftConstSub(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseSubOp());
  CheckCudaError("CudaPerformLeftConstSub");
}

void CudaPerformLeftConstDiv(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseDivOp());
  CheckCudaError("CudaPerformLeftConstDiv");
}

void CudaPerformNormAddOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnCol");
}

void CudaPerformNormSubOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnCol");
}

void CudaPerformNormMultOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnCol");
}

void CudaPerformNormDivOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnCol");
}

void CudaPerformNormAddOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnRow");
}

void CudaPerformNormSubOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnRow");
}

void CudaPerformNormMultOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnRow");
}

void CudaPerformNormDivOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnRow");
}

void CudaPerformReductionSumOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnCol");
}

void CudaPerformReductionMaxOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnCol");
}

void CudaPerformReductionSumOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnRow");
}

void CudaPerformReductionMaxOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnRow");
}

void CudaPerformMaxIndexOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformMaxIndexOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnCol");
}

void CudaPerformMaxIndexOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformMaxIndexOnRowKernel << <block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnRow");
}

void CudaPerformElewiseExp(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, ExpOp());
  CheckCudaError("CudaPerformEleWiseExp");
}

void CudaPerformElewiseLn(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, LnOp());
  CheckCudaError("CudaPerformEleWiseLn");
}

void CudaPerformElewiseSigmoid(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, SigmoidOp());
  CheckCudaError("CudaPerformEleWiseSigmoid");
}

void CudaPerformElewiseNegative(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, NegativeOp());
  CheckCudaError("CudaPerformEleWiseNegative");
}

void CudaPerformConvForward(float* bottom, float* filter, float* bias, float* top, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_desc;
  cudnnTensor4dDescriptor_t bias_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bias_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_desc, filter_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  float one = 1;
  CUDNN_CALL(hipdnnConvolutionForward(handle, bottom_desc, bottom, filter_desc, filter, conv_desc, top_desc, top, CUDNN_RESULT_NO_ACCUMULATE));
  CUDNN_CALL(cudnnAddTensor4d(handle, CUDNN_ADD_SAME_C, &one, bias_desc, bias, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bias_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformConvBackwardData(float* top_diff, float* filter, float* bottom_diff, int num_images, int bottom_num_channels, int top_num_channels, int top_height, int top_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_diff_desc;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_diff_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, (top_height - 1) * stride_vertical + filter_height - 2 * pad_height, (top_width - 1) * stride_horizontal + filter_width - 2 * pad_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_diff_desc, filter_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  CUDNN_CALL(hipdnnConvolutionBackwardData(handle, filter_desc, filter, top_diff_desc, top_diff, conv_desc, bottom_diff_desc, bottom_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_diff_desc));
}

void CudaPerformConvBackwardFilter(float* bottom, float* top_diff, float* filter_diff, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_diff_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_diff_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetFilterDescriptor(filter_diff_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(cudnnSetConvolutionDescriptor(conv_desc, bottom_desc, filter_diff_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle, bottom_desc, bottom, top_diff_desc, top_diff, conv_desc, filter_diff_desc, filter_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_diff_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformConvBackwardBias(float* top_diff, float* bias_diff, int num_images, int top_num_channels, int top_height, int top_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bias_diff_desc;
  cudnnTensor4dDescriptor_t top_diff_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bias_diff_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  CUDNN_CALL(hipdnnConvolutionBackwardBias(handle, top_diff_desc, top_diff, bias_diff_desc, bias_diff, CUDNN_RESULT_NO_ACCUMULATE));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_diff_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bias_diff_desc));
}

void CudaPerformInstanceSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformChannelSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformInstanceSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, desc, top, desc, diff, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformChannelSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, desc, top, desc, diff, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformSigmoidForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_SIGMOID, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformReluForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_RELU, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformTanhForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_TANH, desc, bottom, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformSigmoidBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_SIGMOID, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformReluBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_RELU, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformTanhBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_TANH, desc, top, desc, top_diff, desc, bottom, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(desc));
}

void CudaPerformMaxPoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, (bottom_height - window_height) / stride_vertical + 1, (bottom_width - window_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, bottom_desc, bottom, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, CUDNN_POOLING_AVERAGE, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, (bottom_height - window_height) / stride_vertical + 1, (bottom_width - window_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, bottom_desc, bottom, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformMaxPoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, (bottom_height - window_height) / stride_vertical + 1, (bottom_width - window_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, top_desc, top, top_desc, top_diff, bottom_desc, bottom, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, hipStream_t stream, hipdnnHandle_t handle) {
  cudnnTensor4dDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  cudnnTensor4dDescriptor_t top_desc;

  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(cudnnCreateTensor4dDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(cudnnSetPoolingDescriptor(pool_desc, CUDNN_POOLING_AVERAGE, window_height, window_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, (bottom_height - window_height) / stride_vertical + 1, (bottom_width - window_width) / stride_horizontal + 1));

  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, top_desc, top, top_desc, top_diff, bottom_desc, bottom, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(cudnnDestroyTensor4dDescriptor(bottom_desc));
}

}  // namespace cuda
}  // namespace minerva

